#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "fps_gpu.h"


__device__ void __update(float *__restrict__ dists, int *__restrict__ dists_i, int idx1, int idx2){
    const float v1 = dists[idx1], v2 = dists[idx2];
    const int i1 = dists_i[idx1], i2 = dists_i[idx2];
    dists[idx1] = max(v1, v2);
    dists_i[idx1] = v2 > v1 ? i2 : i1;
}

template <unsigned int block_size>  // 因为下面定义数组的时候要用这个变量所以这里必须定义成模板函数
__global__ void farthest_point_sampling_kernel(int b, int n, int m,
    const float *__restrict__ dataset, float *__restrict__ temp, int *__restrict__ idxs) {
    // dataset: (B, N, 3) 输入点
    // temp: (B, N)
    // output:
    //      idx: (B, M)  每个batch采样m个点

    if (m <= 0) return;

    // block_size == n_thread
    // 每个线程有一个用来存放结果的位置
    __shared__ float dists[block_size];
    __shared__ int dists_i[block_size];

    // 每个block处理一个batch
    int batch_index = blockIdx.x;

    // 直接加上offset把指针推到该block的batch对应的数据处
    dataset += batch_index * n * 3;
    temp += batch_index * n;
    idxs += batch_index * m;

    int tid = threadIdx.x;
    const int stride = block_size;

    // 不随机取初始点而是直接取第0个点
    int old = 0;
    if (threadIdx.x == 0)
        idxs[0] = old;

    __syncthreads();
    for (int j = 1; j < m; j++) {
        int besti = 0;
        float best = -1;
        float x1 = dataset[old * 3 + 0];
        float y1 = dataset[old * 3 + 1];
        float z1 = dataset[old * 3 + 2];

        // 一共有block_size个线程在并行所以每个线程只需要每stride=block_size个点处理一个
        for (int k = tid; k < n; k += stride) {
            float x2, y2, z2;
            x2 = dataset[k * 3 + 0];
            y2 = dataset[k * 3 + 1];
            z2 = dataset[k * 3 + 2];
            // float mag = (x2 * x2) + (y2 * y2) + (z2 * z2);
            // if (mag <= 1e-3)
            // continue;

            float d = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
            
            // 如果到其他采样点的距离更近就采用更近的距离
            float d2 = min(d, temp[k]);
            temp[k] = d2;

            besti = d2 > best ? k : besti;
            best = d2 > best ? d2 : best;
        }
        dists[tid] = best;
        dists_i[tid] = besti;
        __syncthreads();

        // 整理数据，将最大的距离和最大距离对应点的idx逐步存储到dists[0]和dists_i[0]中
        // 每轮只需要一半的线程整理数据
        if (block_size >= 1024) {
            if (tid < 512) {
                __update(dists, dists_i, tid, tid + 512);
            }
            __syncthreads();
        }

        if (block_size >= 512) {
            if (tid < 256) {
                __update(dists, dists_i, tid, tid + 256);
            }
            __syncthreads();
        }
        if (block_size >= 256) {
            if (tid < 128) {
                __update(dists, dists_i, tid, tid + 128);
            }
            __syncthreads();
        }
        if (block_size >= 128) {
            if (tid < 64) {
                __update(dists, dists_i, tid, tid + 64);
            }
            __syncthreads();
        }
        if (block_size >= 64) {
            if (tid < 32) {
                __update(dists, dists_i, tid, tid + 32);
            }
            __syncthreads();
        }
        if (block_size >= 32) {
            if (tid < 16) {
                __update(dists, dists_i, tid, tid + 16);
            }
            __syncthreads();
        }
        if (block_size >= 16) {
            if (tid < 8) {
                __update(dists, dists_i, tid, tid + 8);
            }
            __syncthreads();
        }
        if (block_size >= 8) {
            if (tid < 4) {
                __update(dists, dists_i, tid, tid + 4);
            }
            __syncthreads();
        }
        if (block_size >= 4) {
            if (tid < 2) {
                __update(dists, dists_i, tid, tid + 2);
            }
            __syncthreads();
        }
        if (block_size >= 2) {
            if (tid < 1) {
                __update(dists, dists_i, tid, tid + 1);
            }
            __syncthreads();
        }

        // 将最大距离对应点的idx存到idxs中
        // 只需要一个线程来做就可以了
        old = dists_i[0];
        if (tid == 0)
            idxs[j] = old;
    }
}

void farthest_point_sampling_kernel_launcher(int b, int n, int m,
    const float *dataset, float *temp, int *idxs, hipStream_t stream) {
    // dataset: (B, N, 3)
    // tmp: (B, N)
    // output:
    //      idx: (B, M)

    hipError_t err;
    unsigned int n_threads = opt_n_threads(n);

    switch (n_threads) {
    case 1024:
        farthest_point_sampling_kernel<1024><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 512:
        farthest_point_sampling_kernel<512><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 256:
        farthest_point_sampling_kernel<256><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 128:
        farthest_point_sampling_kernel<128><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 64:
        farthest_point_sampling_kernel<64><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 32:
        farthest_point_sampling_kernel<32><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 16:
        farthest_point_sampling_kernel<16><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 8:
        farthest_point_sampling_kernel<8><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 4:
        farthest_point_sampling_kernel<4><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 2:
        farthest_point_sampling_kernel<2><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    case 1:
        farthest_point_sampling_kernel<1><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
    default:
        farthest_point_sampling_kernel<512><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    }

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
